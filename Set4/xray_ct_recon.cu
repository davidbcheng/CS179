#include "hip/hip_runtime.h"

/* 
Based off work by Nelson, et al.
Brigham Young University (2010)

Adapted by Kevin Yuh (2015)
*/


#include <stdio.h>
#include <hip/hip_runtime.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hipfft/hipfft.h>

#define PI 3.14159265358979


/* Check errors on CUDA runtime functions */
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess) 
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        exit(code);
    }
}



/* Check errors on cuFFT functions */
void gpuFFTchk(int errval){
    if (errval != HIPFFT_SUCCESS){
        printf("Failed FFT call, error code %d\n", errval);
    }
}


/* Check errors on CUDA kernel calls */
void checkCUDAKernelError()
{
    hipError_t err = hipGetLastError();
    if  (hipSuccess != err){
        fprintf(stderr, "Error %s\n", hipGetErrorString(err));
    } else {
        fprintf(stderr, "No kernel error detected\n");
    }

}

__global__
void
cudaHighPassKernel(hipfftComplex *raw_data, int length) {
    // Determine the index of the output data we are writing to by
    // the block id and the thread id
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

    while(index < length)
    {
        raw_data[index].x = fabsf(1 - (2 * index) / length);
        raw_data[index].y = fabsf(1 - (2 * index) / length);

        index += blockDim.x + gridDim.x;
    }
}

__global__
void
cudaBackProjectionKernel(float *sinogram, float *result, int size_result,
    int nAngles, int sinogram_width, int side_length) {

    int x_p = threadIdx.x + blockDim.x * blockIdx.x;
    int y_p = threadIdx.y + blockDim.y * blockIdx.y;
    float x_0 = (float) x_p - (side_length / 2);
    float y_0 = (side_length / 2) - (float) y_p;

    int angle;

    for (angle = 0; angle < nAngles; ++angle)
    {
        float theta = ((float) angle / nAngles) * PI;
        float d;
        if (theta == 0)
        {
            d = x_0;
        }
        else if (theta == PI / 2)
        {
            d = y_0;
        }
        else
        {
            float m = -1 * cosf(theta) / sinf(theta);
            float q = -1 / m;

            float x_1 = (y_0 - m * x_0) / (q - m);
            float y_1 = q * x_1;
            d = sqrtf(x_1 * x_1 + y_1 * y_1);

            if (x_1 < 0 || (q < 0 && x_1 > 0))
            {
                d = -1 * d;
            }
        }

        result[x_p + side_length * y_p] += sinogram[(int) d + nAngles * angle];
    }
}

int main(int argc, char** argv){

    if (argc != 7){
        fprintf(stderr, "Incorrect number of arguments.\n\n");
        fprintf(stderr, "\nArguments: \n \
        < Sinogram filename > \n \
        < Width or height of original image, whichever is larger > \n \
        < Number of angles in sinogram >\n \
        < threads per block >\n \
        < number of blocks >\n \
        < output filename >\n");
        exit(EXIT_FAILURE);
    }


    /********** Parameters **********/

    int width = atoi(argv[2]);
    int height = width;
    int sinogram_width = (int)ceilf( height * sqrt(2) );

    int nAngles = atoi(argv[3]);


    int threadsPerBlock = atoi(argv[4]);
    int nBlocks = atoi(argv[5]);


    /********** Data storage *********/


    // GPU DATA STORAGE
    hipfftComplex *dev_sinogram_cmplx;
    float *dev_sinogram_float; 
    float* output_dev;  // Image storage


    hipfftComplex *sinogram_host;

    size_t size_result = width*height*sizeof(float);
    float *output_host = (float *)malloc(size_result);




    /*********** Set up IO, Read in data ************/

    sinogram_host = (hipfftComplex *)malloc(  sinogram_width*nAngles*sizeof(hipfftComplex) );

    FILE *dataFile = fopen(argv[1],"r");
    if (dataFile == NULL){
        fprintf(stderr, "Sinogram file missing\n");
        exit(EXIT_FAILURE);
    }

    FILE *outputFile = fopen(argv[6], "w");
    if (outputFile == NULL){
        fprintf(stderr, "Output file cannot be written\n");
        exit(EXIT_FAILURE);
    }

    int j, i;

    for(i = 0; i < nAngles * sinogram_width; i++){
        fscanf(dataFile,"%f",&sinogram_host[i].x);
        sinogram_host[i].y = 0;
    }

    fclose(dataFile);


    /*********** Assignment starts here *********/

    /* TODO: Allocate memory for all GPU storage above, copy input sinogram
    over to dev_sinogram_cmplx. */

    hipMalloc((void **) &dev_sinogram_cmplx, sinogram_width*nAngles*sizeof(hipfftComplex));
    hipMalloc((void **) &output_dev, size_result);
    hipMalloc((void **) &dev_sinogram_float, sinogram_width*nAngles*sizeof(float));

    hipMemcpy(dev_sinogram_float, sinogram_host,
        sizeof(hipfftComplex) * sinogram_width * nAngles, hipMemcpyHostToDevice);



    /* TODO 1: Implement the high-pass filter:
        - Use cuFFT for the forward FFT
        - Create your own kernel for the frequency scaling.
        - Use cuFFT for the inverse FFT
        - extract real components to floats
        - Free the original sinogram (dev_sinogram_cmplx)

        Note: If you want to deal with real-to-complex and complex-to-real
        transforms in cuFFT, you'll have to slightly change our code above.
    */

    hipfftHandle plan;
    hipfftHandle plan2;
    int batch = nAngles;
    hipfftPlan1d(&plan, sinogram_width*nAngles*sizeof(hipfftComplex),
        HIPFFT_C2C, batch);
    hipfftPlan1d(&plan2, sinogram_width*nAngles*sizeof(hipfftComplex),
        HIPFFT_C2R, batch);


    hipfftExecC2C(plan, dev_sinogram_cmplx, dev_sinogram_cmplx, HIPFFT_FORWARD);

    cudaHighPassKernel<<<nBlocks, threadsPerBlock>>> (dev_sinogram_cmplx,
        sinogram_width*nAngles*sizeof(hipfftComplex));

    hipfftExecC2R(plan2, dev_sinogram_cmplx, dev_sinogram_float, HIPFFT_BACKWARD);
    hipfftDestroy(plan);
    hipfftDestroy(plan2);

    hipFree(dev_sinogram_cmplx);


    /* TODO 2: Implement backprojection.
        - Allocate memory for the output image.
        - Create your own kernel to accelerate backprojection.
        - Copy the reconstructed image back to output_host.
        - Free all remaining memory on the GPU.
    */

    hipMalloc((void **) &output_dev, size_result);
    cudaBackProjectionKernel <<<nBlocks, threadsPerBlock>>> (dev_sinogram_float,
     output_dev, size_result, nAngles, sinogram_width, height);
    hipMemcpy(output_host, output_dev, size_result, hipMemcpyDeviceToHost);

    hipFree(dev_sinogram_float);
    hipFree(output_dev);

    /* Export image data. */

    for(j = 0; j < width; j++){
        for(i = 0; i < height; i++){
            fprintf(outputFile, "%e ",output_host[j*width + i]);
        }
        fprintf(outputFile, "\n");
    }


    /* Cleanup: Free host memory, close files. */

    free(sinogram_host);
    free(output_host);

    fclose(outputFile);

    return 0;
}



